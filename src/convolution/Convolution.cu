#include "hip/hip_runtime.h"
//
// Created by giova on 31/05/2022.
//

#include "Convolution.cuh"
#include "kernel/Kernel.h"

extern __constant__ float MASK[MASK_WIDTH * MASK_WIDTH];

// number of input elements per block
#define w (TILE_WIDTH + MASK_WIDTH - 1)


__global__ void convolutionNaive(const float* __restrict__ data, const float* __restrict__ mask, float* result,
                                 int width, int height, int channels) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        float accum;

        for (int k = 0; k < channels; k++){
            accum = 0;
            for (int y = -MASK_RADIUS; y <= MASK_RADIUS; y++) {
                for (int x = -MASK_RADIUS; x <= MASK_RADIUS; x++) {
                    if((row + y) > -1 && (row + y) < height && (col + x) > -1 && (col + x) < width) {
                        accum += data[((row + y) * width + col + x) * channels + k] *
                                 mask[(y + MASK_RADIUS) * MASK_WIDTH + x + MASK_RADIUS];
                    }
                }
            }
            result[(row * width + col) * channels + k] = accum;
        }
    }
}

__global__ void convolutionNaiveNoPadding(const float* __restrict__ data, const float* __restrict__ mask, float* result,
                                 int width, int height, int channels) {
    int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (col < width - 2 && row < height - 2) {
        float accum;

        for (int k = 0; k < channels; k++){
            accum = 0;
            for (int y = -MASK_RADIUS; y <= MASK_RADIUS; y++) {
                for (int x = -MASK_RADIUS; x <= MASK_RADIUS; x++) {
                    accum += data[((row + y) * width + col + x) * channels + k] *
                             mask[(y + MASK_RADIUS) * MASK_WIDTH + x + MASK_RADIUS];
                }
            }
            result[(row * (width - 2) + col) * channels + k] = accum;
        }
    }
}

__global__ void convolutionConstantMemory(const float* __restrict__ data, float* result,
                                 int width, int height, int channels) {
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    if (col < width && row < height) {
        float accum;

        for (int k = 0; k < channels; k++){
            accum = 0;
            for (int y = -MASK_RADIUS; y <= MASK_RADIUS; y++) {
                for (int x = -MASK_RADIUS; x <= MASK_RADIUS; x++) {
                    if((row + y) > -1 && (row + y) < height && (col + x) > -1 && (col + x) < width) {
                        accum += data[((row + y) * width + col + x) * channels + k] * MASK[(y + MASK_RADIUS) * MASK_WIDTH + x + MASK_RADIUS];
                    }
                }
            }
            result[(row * width + col) * channels + k] = accum;
        }
    }
}

__global__ void convolutionConstantMemoryNoPadding(const float* __restrict__ data, float* result,
                                          int width, int height, int channels) {
    int col = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int row = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (col < width - 2 && row < height - 2) {
        float accum;

        for (int k = 0; k < channels; k++){
            accum = 0;
            for (int y = -MASK_RADIUS; y <= MASK_RADIUS; y++) {
                for (int x = -MASK_RADIUS; x <= MASK_RADIUS; x++) {
                    accum += data[((row + y) * width + col + x) * channels + k] * MASK[(y + MASK_RADIUS) * MASK_WIDTH + x + MASK_RADIUS];
                }
            }
            result[(row * (width - 2) + col) * channels + k] = accum;
        }
    }
}

__global__ void convolutionTiling(const float* __restrict__ data, float* result,
                                          int width, int height, int channels) {
    __shared__ float data_ds[w][w];

    for (int k = 0; k < channels; k++) {
        // First batch loading
        int dest = threadIdx.y * TILE_WIDTH + threadIdx.x;
        int destY = dest / w;
        int destX = dest % w;
        int srcY = blockIdx.y * TILE_WIDTH + destY - MASK_RADIUS;
        int srcX = blockIdx.x * TILE_WIDTH + destX - MASK_RADIUS;
        int src = (srcY * width + srcX) * channels + k;
        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width) {
            data_ds[destY][destX] = data[src];
        } else {
            data_ds[destY][destX] = 0;
        }

        // Second batch loading
        dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
        destY = dest / w;
        destX = dest % w;
        srcY = blockIdx.y * TILE_WIDTH + destY - MASK_RADIUS;
        srcX = blockIdx.x * TILE_WIDTH + destX - MASK_RADIUS;
        src = (srcY * width + srcX) * channels + k;
        if (destY < w) {
            if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width) {
                data_ds[destY][destX] = data[src];
            } else {
                data_ds[destY][destX] = 0;
            }
        }
        __syncthreads();

        float accum = 0;
        for (int y = 0; y < MASK_WIDTH; y++) {
            for (int x = 0; x < MASK_WIDTH; x++) {
                accum += data_ds[threadIdx.y + y][threadIdx.x + x]
                         * MASK[y * MASK_WIDTH + x];
            }
        }
        int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
        int x = blockIdx.x * TILE_WIDTH + threadIdx.x;
        if (y < height && x < width)
            result[(y * width + x) * channels + k] = accum;
        __syncthreads();
    }
}

__global__ void convolutionTilingNoPadding(const float* __restrict__ data, float* result,
                                  int width, int height, int channels) {
    __shared__ float data_ds[w][w];

    for (int k = 0; k < channels; k++) {
        // First batch loading
        int dest = threadIdx.y * TILE_WIDTH + threadIdx.x;
        int destY = dest / w;
        int destX = dest % w;
        int srcY = blockIdx.y * TILE_WIDTH + destY - MASK_RADIUS + 1;
        int srcX = blockIdx.x * TILE_WIDTH + destX - MASK_RADIUS + 1;
        int src = (srcY * width + srcX) * channels + k;
        if (srcY >= 0 && srcY < height - 2 && srcX >= 0 && srcX < width - 2) {
            data_ds[destY][destX] = data[src];
        } else {
            data_ds[destY][destX] = 255;
        }

        // Second batch loading
        dest = threadIdx.y * TILE_WIDTH + threadIdx.x + TILE_WIDTH * TILE_WIDTH;
        destY = dest / w;
        destX = dest % w;
        srcY = blockIdx.y * TILE_WIDTH + destY - MASK_RADIUS + 1;
        srcX = blockIdx.x * TILE_WIDTH + destX - MASK_RADIUS + 1;
        src = (srcY * width + srcX) * channels + k;
        if (destY < w) {
            if (srcY >= 0 && srcY < height - 2 && srcX >= 0 && srcX < width - 2) {
                data_ds[destY][destX] = data[src];
            } else {
                data_ds[destY][destX] = 255;
            }
        }
        __syncthreads();

        float accum = 0;
        for (int y = 0; y < MASK_WIDTH; y++) {
            for (int x = 0; x < MASK_WIDTH; x++) {
                accum += data_ds[threadIdx.y + y][threadIdx.x + x]
                         * MASK[y * MASK_WIDTH + x];
            }
        }
        int y = blockIdx.y * TILE_WIDTH + threadIdx.y;
        int x = blockIdx.x * TILE_WIDTH + threadIdx.x;
        if (y < (height - 2) && x < (width - 2))
            result[(y * (width - 2) + x) * channels + k] = accum;
        __syncthreads();
    }
}