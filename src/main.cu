#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <string>
#include <cassert>

#include "image/PpmParser.h"
#include "kernel/Kernel.h"
#include "convolution/Convolution.cuh"

// From terminal
#define IMPORT_PATH "resources/source/"
#define EXPORT_PATH "resources/results/"

// From IDE
//#define IMPORT_PATH "../resources/source/"
//#define EXPORT_PATH "../resources/results/"

#define IMAGE "lake"

#define BLOCK_WIDTH_NAIVE 8

#define BLOCK_WIDTH (TILE_WIDTH)
static_assert(BLOCK_WIDTH * BLOCK_WIDTH <= 1024, "max number of threads per block exceeded");

#define ITER 1

#define SOA true
#define AOS false

#define ASYNC true

#define NAIVE false
#define TILING true

__constant__ float MASK[MASK_WIDTH * MASK_WIDTH];


static void CheckCudaErrorAux(const char *, unsigned, const char *,
                              hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
                              const char *statement, hipError_t err) {
    if (err == hipSuccess)
        return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "("
              << err << ") at " << file << ":" << line << std::endl;
    exit(1);
}

int main() {
    assert(MASK_WIDTH == 3);

    std::string filename;
    std::string output_name;

    std::string log;
    log.append("Cuda version ");

    filename.append(IMPORT_PATH).append(IMAGE).append(".ppm");
    output_name.append(EXPORT_PATH).append(IMAGE).append("Cuda");

    float *kernel = createKernel(kernelsType::outline);

    float time = 0;

    std::chrono::high_resolution_clock::time_point startTime;
    std::chrono::high_resolution_clock::time_point endTime;

    if (AOS) {
        output_name.append("AoS");
        log.append("AoS ");

        Image_t* inputImage = PPM_import(filename.c_str());

        int imageWidth = image_getWidth(inputImage);
        int imageHeight = image_getHeight(inputImage);
        int imageChannels = image_getChannels(inputImage);

        Image_t* outputImage;

        float *imageData;
        float *outputData;

        int outputWidth = imageWidth - 2;
        int outputHeight = imageHeight - 2;

        if (NAIVE) {
            log.append("naive ");

            float *device_imageData;
            float *device_outputData;
            float *device_maskData;

            for (int i = 0; i < ITER; i++) {
                if (i != 0) image_delete(outputImage);

                outputImage = new_image(outputWidth, outputHeight, imageChannels);

                imageData = image_getData(inputImage);
                outputData = image_getData(outputImage);

                startTime = std::chrono::high_resolution_clock::now();

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageData,
                                             imageWidth * imageHeight * imageChannels * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputData,
                                             outputWidth * outputHeight * imageChannels * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_maskData,
                                             MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                CUDA_CHECK_RETURN(hipMemcpy(device_imageData, imageData,
                                             imageWidth * imageHeight * imageChannels * sizeof(float),
                                             hipMemcpyHostToDevice));
                CUDA_CHECK_RETURN(hipMemcpy(device_maskData, kernel,
                                             MASK_WIDTH * MASK_WIDTH * sizeof(float),
                                             hipMemcpyHostToDevice));

                dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                             ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                convolutionNaiveNoPadding<<<dimGrid, dimBlock>>>(device_imageData, device_maskData,
                                                                 device_outputData, imageWidth, imageHeight,
                                                                 imageChannels);

                CUDA_CHECK_RETURN(hipDeviceSynchronize());

                CUDA_CHECK_RETURN(hipMemcpy(outputData, device_outputData,
                                             outputWidth * outputHeight * imageChannels * sizeof(float),
                                             hipMemcpyDeviceToHost));

                endTime = std::chrono::high_resolution_clock::now();
                time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                hipFree(device_imageData);
                hipFree(device_outputData);
                hipFree(device_maskData);
            }
        }

        if (TILING) {
            log.append("with tiling ");
            output_name.append("Tiling");

            float *device_imageData;
            float *device_outputData;

            for (int i = 0; i < ITER; i++) {
                if (i != 0) image_delete(outputImage);

                outputImage = new_image(outputWidth, outputHeight, imageChannels);

                imageData = image_getData(inputImage);
                outputData = image_getData(outputImage);

                startTime = std::chrono::high_resolution_clock::now();

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageData,
                                             imageWidth * imageHeight * imageChannels * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputData,
                                             outputWidth * outputHeight * imageChannels * sizeof(float)));
                CUDA_CHECK_RETURN(hipMemcpy(device_imageData, imageData,
                                             imageWidth * imageHeight * imageChannels * sizeof(float),
                                             hipMemcpyHostToDevice));
                CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH),
                             ceil((float) imageHeight / BLOCK_WIDTH));
                dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);

                convolutionTilingNoPadding<<<dimGrid, dimBlock>>>(device_imageData,
                                                                  device_outputData, imageWidth, imageHeight,
                                                                  imageChannels);

                CUDA_CHECK_RETURN(hipMemcpy(outputData, device_outputData,
                                             outputWidth * outputHeight * imageChannels * sizeof(float),
                                             hipMemcpyDeviceToHost));

                endTime = std::chrono::high_resolution_clock::now();
                time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                hipFree(device_imageData);
                hipFree(device_outputData);

                hipFree(MASK);
            }
        }
        log.append("took ").append(std::to_string(time/ITER)).append(" seconds");
        printf("%s\n", log.c_str());

        output_name.append(".ppm");

        PPM_export(output_name.c_str(), outputImage);

        image_delete(outputImage);
        image_delete(inputImage);
    }

    if(SOA) {
        output_name.append("SoA");
        log.append("SoA ");

        ImageSoA_t* inputImage = PPM_importSoA(filename.c_str());

        int imageWidth = image_getWidth(inputImage);
        int imageHeight = image_getHeight(inputImage);
        int imageChannels = image_getChannels(inputImage);

        assert(imageChannels == 3);

        float *imageDataR;
        float *imageDataG;
        float *imageDataB;

        ImageSoA_t* outputImage;

        float *outputDataR;
        float *outputDataG;
        float *outputDataB;

        float *device_imageDataR;
        float *device_imageDataG;
        float *device_imageDataB;

        float *device_outputDataR;
        float *device_outputDataG;
        float *device_outputDataB;

        int outputWidth = imageWidth - 2;
        int outputHeight = imageHeight - 2;

        if (NAIVE && !ASYNC) {
            log.append("naive ");
            output_name.append("Naive");

            float *device_maskData;

            for (int i = 0; i < ITER; i++) {
                if (i != 0) image_delete(outputImage);

                outputImage = new_imageSoA(outputWidth, outputHeight, imageChannels);

                imageDataR = image_getR(inputImage);
                imageDataG = image_getG(inputImage);
                imageDataB = image_getB(inputImage);

                outputDataR = image_getR(outputImage);
                outputDataG = image_getG(outputImage);
                outputDataB = image_getB(outputImage);

                startTime = std::chrono::high_resolution_clock::now();

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataR,
                                             imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataG,
                                             imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataB,
                                             imageWidth * imageHeight * sizeof(float)));

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataR,
                                             outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataG,
                                             outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataB,
                                             outputWidth * outputHeight * sizeof(float)));

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_maskData,
                                             MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                CUDA_CHECK_RETURN(hipMemcpy(device_imageDataR, imageDataR,
                                             imageWidth * imageHeight * sizeof(float),
                                             hipMemcpyHostToDevice));
                CUDA_CHECK_RETURN(hipMemcpy(device_imageDataG, imageDataG,
                                             imageWidth * imageHeight * sizeof(float),
                                             hipMemcpyHostToDevice));
                CUDA_CHECK_RETURN(hipMemcpy(device_imageDataB, imageDataB,
                                             imageWidth * imageHeight * sizeof(float),
                                             hipMemcpyHostToDevice));

                CUDA_CHECK_RETURN(hipMemcpy(device_maskData, kernel,
                                             MASK_WIDTH * MASK_WIDTH * sizeof(float),
                                             hipMemcpyHostToDevice));

                dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                             ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                convolutionNaiveNoPaddingSoA<<<dimGrid, dimBlock>>>(device_imageDataR, device_imageDataG, device_imageDataB,
                                                           device_maskData, device_outputDataR, device_outputDataG,
                                                           device_outputDataB,
                                                           imageWidth, imageHeight);

                CUDA_CHECK_RETURN(hipDeviceSynchronize());

                CUDA_CHECK_RETURN(hipMemcpy(outputDataR, device_outputDataR,
                                             outputWidth * outputHeight * sizeof(float),
                                             hipMemcpyDeviceToHost));
                CUDA_CHECK_RETURN(hipMemcpy(outputDataG, device_outputDataG,
                                             outputWidth * outputHeight * sizeof(float),
                                             hipMemcpyDeviceToHost));
                CUDA_CHECK_RETURN(hipMemcpy(outputDataB, device_outputDataB,
                                             outputWidth * outputHeight * sizeof(float),
                                             hipMemcpyDeviceToHost));

                endTime = std::chrono::high_resolution_clock::now();
                time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                hipFree(device_imageDataR);
                hipFree(device_imageDataG);
                hipFree(device_imageDataB);
                hipFree(device_outputDataR);
                hipFree(device_outputDataG);
                hipFree(device_outputDataB);

                hipFree(device_maskData);
            }
        }

        if(NAIVE && ASYNC) {
            log.append("naive with async loading ");
            output_name.append("NaiveAsync");

            float *device_maskData;

            for (int i = 0; i < ITER; i++) {
                if (i != 0) image_delete(outputImage);

                outputImage = new_imageSoA(outputWidth, outputHeight, imageChannels);

                hipStream_t stream1;
                hipStream_t stream2;
                hipStream_t stream3;

                hipStreamCreate(&stream1);
                hipStreamCreate(&stream2);
                hipStreamCreate(&stream3);

                float* host_imageDataR;
                float* host_imageDataG;
                float* host_imageDataB;

                float* host_outputDataR;
                float* host_outputDataG;
                float* host_outputDataB;

                imageDataR = image_getR(inputImage);
                imageDataG = image_getG(inputImage);
                imageDataB = image_getB(inputImage);

                outputDataR = image_getR(outputImage);
                outputDataG = image_getG(outputImage);
                outputDataB = image_getB(outputImage);

                startTime = std::chrono::high_resolution_clock::now();

                dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                             ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_maskData,
                                             MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                CUDA_CHECK_RETURN(hipMemcpy(device_maskData, kernel,
                                             MASK_WIDTH * MASK_WIDTH * sizeof(float),
                                             hipMemcpyHostToDevice));

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataR,
                                             imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataG,
                                             imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataB,
                                             imageWidth * imageHeight * sizeof(float)));

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataR,
                                             outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataG,
                                             outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataB,
                                             outputWidth * outputHeight * sizeof(float)));

                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_imageDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_imageDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_imageDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_outputDataR,
                                                 outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_outputDataG,
                                                 outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_outputDataB,
                                                 outputWidth * outputHeight * sizeof(float)));

//                NON DOVREBBE ESSERE ASYNC VISTO CHE FACCIO CON MEMORIA PINNED E CON MEMORIA NON!! PINNED
                CUDA_CHECK_RETURN(hipMemcpyAsync(host_imageDataR, imageDataR, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToHost, stream1));
                CUDA_CHECK_RETURN(hipMemcpyAsync(host_imageDataG, imageDataG, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToHost, stream2));
                CUDA_CHECK_RETURN(hipMemcpyAsync(host_imageDataB, imageDataB, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToHost, stream3));

                CUDA_CHECK_RETURN(hipMemcpyAsync(device_imageDataR, host_imageDataR, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToDevice, stream1));
                CUDA_CHECK_RETURN(hipMemcpyAsync(device_imageDataG, host_imageDataG, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToDevice, stream2));
                CUDA_CHECK_RETURN(hipMemcpyAsync(device_imageDataB, host_imageDataB, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToDevice, stream3));

                convolutionNaiveNoPaddingSoAChannelR<<<dimGrid, dimBlock, 0, stream1>>>(device_imageDataR, device_maskData, device_outputDataR,
                                                                                        imageWidth, imageHeight);

                convolutionNaiveNoPaddingSoAChannelG<<<dimGrid, dimBlock, 0, stream2>>>(device_imageDataG, device_maskData, device_outputDataG,
                                                                                        imageWidth, imageHeight);

                convolutionNaiveNoPaddingSoAChannelB<<<dimGrid, dimBlock, 0, stream3>>>(device_imageDataB, device_maskData, device_outputDataB,
                                                                                        imageWidth, imageHeight);

                CUDA_CHECK_RETURN(hipMemcpyAsync(host_outputDataR, device_outputDataR,
                                                  outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyDeviceToHost, stream1));

                CUDA_CHECK_RETURN(hipMemcpyAsync(host_outputDataG, device_outputDataG,
                                                  outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyDeviceToHost, stream2));

                CUDA_CHECK_RETURN(hipMemcpyAsync(host_outputDataB, device_outputDataB,
                                                  outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyDeviceToHost, stream3));

                CUDA_CHECK_RETURN(hipMemcpyAsync(outputDataR, host_outputDataR, outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyHostToHost, stream1));
                CUDA_CHECK_RETURN(hipMemcpyAsync(outputDataG, host_outputDataG, outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyHostToHost, stream2));
                CUDA_CHECK_RETURN(hipMemcpyAsync(outputDataB, host_outputDataB, outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyHostToHost, stream3));

                CUDA_CHECK_RETURN(hipDeviceSynchronize());

                endTime = std::chrono::high_resolution_clock::now();
                time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                hipStreamDestroy(stream1);
                hipStreamDestroy(stream2);
                hipStreamDestroy(stream3);

                hipFree(device_imageDataR);
                hipFree(device_imageDataG);
                hipFree(device_imageDataB);
                hipFree(device_outputDataR);
                hipFree(device_outputDataG);
                hipFree(device_outputDataB);

                hipFree(device_maskData);

                hipHostFree(host_imageDataR);
                hipHostFree(host_imageDataG);
                hipHostFree(host_imageDataB);

                hipHostFree(host_outputDataR);
                hipHostFree(host_outputDataG);
                hipHostFree(host_outputDataB);
            }
        }


        if (TILING && !ASYNC) {
            log.append("tiling ");
            output_name.append("Tiling");

            for (int i = 0; i < ITER; i++) {
                if (i != 0) image_delete(outputImage);

                outputImage = new_imageSoA(outputWidth, outputHeight, imageChannels);

                imageDataR = image_getR(inputImage);
                imageDataG = image_getG(inputImage);
                imageDataB = image_getB(inputImage);

                outputDataR = image_getR(outputImage);
                outputDataG = image_getG(outputImage);
                outputDataB = image_getB(outputImage);

                startTime = std::chrono::high_resolution_clock::now();

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataR,
                                             imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataG,
                                             imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataB,
                                             imageWidth * imageHeight * sizeof(float)));

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataR,
                                             outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataG,
                                             outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataB,
                                             outputWidth * outputHeight * sizeof(float)));

                CUDA_CHECK_RETURN(hipMemcpy(device_imageDataR, imageDataR,
                                             imageWidth * imageHeight * sizeof(float),
                                             hipMemcpyHostToDevice));
                CUDA_CHECK_RETURN(hipMemcpy(device_imageDataG, imageDataG,
                                             imageWidth * imageHeight * sizeof(float),
                                             hipMemcpyHostToDevice));
                CUDA_CHECK_RETURN(hipMemcpy(device_imageDataB, imageDataB,
                                             imageWidth * imageHeight * sizeof(float),
                                             hipMemcpyHostToDevice));

                CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH),
                             ceil((float) imageHeight / BLOCK_WIDTH));
                dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);

                convolutionTilingNoPaddingSoA<<<dimGrid, dimBlock>>>(device_imageDataR, device_imageDataG,
                                                                     device_imageDataB,
                                                                     device_outputDataR, device_outputDataG,
                                                                     device_outputDataB,
                                                                     imageWidth, imageHeight);

                CUDA_CHECK_RETURN(hipMemcpy(outputDataR, device_outputDataR,
                                             outputWidth * outputHeight * sizeof(float),
                                             hipMemcpyDeviceToHost));
                CUDA_CHECK_RETURN(hipMemcpy(outputDataG, device_outputDataG,
                                             outputWidth * outputHeight * sizeof(float),
                                             hipMemcpyDeviceToHost));
                CUDA_CHECK_RETURN(hipMemcpy(outputDataB, device_outputDataB,
                                             outputWidth * outputHeight * sizeof(float),
                                             hipMemcpyDeviceToHost));

                endTime = std::chrono::high_resolution_clock::now();
                time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                hipFree(device_imageDataR);
                hipFree(device_imageDataG);
                hipFree(device_imageDataB);
                hipFree(device_outputDataR);
                hipFree(device_outputDataG);
                hipFree(device_outputDataB);

                hipFree(MASK);
            }
        }

        if(TILING && ASYNC) {
            log.append("tiling with async loading ");
            output_name.append("TilingAsync");

            for (int i = 0; i < ITER; i++) {
                if (i != 0) image_delete(outputImage);

                outputImage = new_imageSoA(outputWidth, outputHeight, imageChannels);

                hipStream_t stream1;
                hipStream_t stream2;
                hipStream_t stream3;

                hipStreamCreate(&stream1);
                hipStreamCreate(&stream2);
                hipStreamCreate(&stream3);

                float* host_imageDataR;
                float* host_imageDataG;
                float* host_imageDataB;

                float* host_outputDataR;
                float* host_outputDataG;
                float* host_outputDataB;

                imageDataR = image_getR(inputImage);
                imageDataG = image_getG(inputImage);
                imageDataB = image_getB(inputImage);

                outputDataR = image_getR(outputImage);
                outputDataG = image_getG(outputImage);
                outputDataB = image_getB(outputImage);

                startTime = std::chrono::high_resolution_clock::now();

                CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH),
                             ceil((float) imageHeight / BLOCK_WIDTH));
                dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataR,
                                             imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataG,
                                             imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataB,
                                             imageWidth * imageHeight * sizeof(float)));

                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataR,
                                             outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataG,
                                             outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataB,
                                             outputWidth * outputHeight * sizeof(float)));

                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_imageDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_imageDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_imageDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_outputDataR,
                                                 outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_outputDataG,
                                                 outputWidth * outputHeight * sizeof(float)));
                CUDA_CHECK_RETURN(hipHostMalloc((void **) &host_outputDataB,
                                                 outputWidth * outputHeight * sizeof(float)));

//                from pageable host memory to pinned host memory
                CUDA_CHECK_RETURN(hipMemcpy(host_imageDataR, imageDataR, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToHost));
                CUDA_CHECK_RETURN(hipMemcpy(host_imageDataG, imageDataG, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToHost));
                CUDA_CHECK_RETURN(hipMemcpy(host_imageDataB, imageDataB, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToHost));

//                CUDA_CHECK_RETURN(hipMemcpyAsync(host_imageDataR, imageDataR, imageWidth * imageHeight * sizeof(float),
//                                             hipMemcpyHostToHost, stream1));
//                CUDA_CHECK_RETURN(hipMemcpyAsync(host_imageDataG, imageDataG, imageWidth * imageHeight * sizeof(float),
//                                             hipMemcpyHostToHost, stream2));
//                CUDA_CHECK_RETURN(hipMemcpyAsync(host_imageDataB, imageDataB, imageWidth * imageHeight * sizeof(float),
//                                             hipMemcpyHostToHost, stream3));

                CUDA_CHECK_RETURN(hipMemcpyAsync(device_imageDataR, host_imageDataR, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToDevice, stream1));
                CUDA_CHECK_RETURN(hipMemcpyAsync(device_imageDataG, host_imageDataG, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToDevice, stream2));
                CUDA_CHECK_RETURN(hipMemcpyAsync(device_imageDataB, host_imageDataB, imageWidth * imageHeight * sizeof(float),
                                                  hipMemcpyHostToDevice, stream3));

                convolutionTilingNoPaddingSoAChannelR<<<dimGrid, dimBlock, 0, stream1>>>(device_imageDataR, device_outputDataR,
                                                                                        imageWidth, imageHeight);

                convolutionTilingNoPaddingSoAChannelG<<<dimGrid, dimBlock, 0, stream2>>>(device_imageDataG, device_outputDataG,
                                                                                        imageWidth, imageHeight);

                convolutionTilingNoPaddingSoAChannelB<<<dimGrid, dimBlock, 0, stream3>>>(device_imageDataB, device_outputDataB,
                                                                                        imageWidth, imageHeight);

                CUDA_CHECK_RETURN(hipMemcpyAsync(host_outputDataR, device_outputDataR,
                                                  outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyDeviceToHost, stream1));

                CUDA_CHECK_RETURN(hipMemcpyAsync(host_outputDataG, device_outputDataG,
                                                  outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyDeviceToHost, stream2));

                CUDA_CHECK_RETURN(hipMemcpyAsync(host_outputDataB, device_outputDataB,
                                                  outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyDeviceToHost, stream3));

                CUDA_CHECK_RETURN(hipDeviceSynchronize());

//                from pinned host memory to pageable host memory
                CUDA_CHECK_RETURN(hipMemcpy(outputDataR, host_outputDataR, outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyHostToHost));
                CUDA_CHECK_RETURN(hipMemcpy(outputDataG, host_outputDataG, outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyHostToHost));
                CUDA_CHECK_RETURN(hipMemcpy(outputDataB, host_outputDataB, outputWidth * outputHeight * sizeof(float),
                                                  hipMemcpyHostToHost));

//                CUDA_CHECK_RETURN(hipMemcpyAsync(outputDataR, host_outputDataR, outputWidth * outputHeight * sizeof(float),
//                                             hipMemcpyHostToHost, stream1));
//                CUDA_CHECK_RETURN(hipMemcpyAsync(outputDataG, host_outputDataG, outputWidth * outputHeight * sizeof(float),
//                                             hipMemcpyHostToHost, stream2));
//                CUDA_CHECK_RETURN(hipMemcpyAsync(outputDataB, host_outputDataB, outputWidth * outputHeight * sizeof(float),
//                                             hipMemcpyHostToHost, stream3));


                endTime = std::chrono::high_resolution_clock::now();
                time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                hipStreamDestroy(stream1);
                hipStreamDestroy(stream2);
                hipStreamDestroy(stream3);

                hipFree(device_imageDataR);
                hipFree(device_imageDataG);
                hipFree(device_imageDataB);
                hipFree(device_outputDataR);
                hipFree(device_outputDataG);
                hipFree(device_outputDataB);

                hipHostFree(host_imageDataR);
                hipHostFree(host_imageDataG);
                hipHostFree(host_imageDataB);

                hipHostFree(host_outputDataR);
                hipHostFree(host_outputDataG);
                hipHostFree(host_outputDataB);

                hipFree(MASK);
            }
        }

        log.append("took ").append(std::to_string(time/ITER)).append(" seconds");
        printf("%s\n", log.c_str());

        output_name.append(".ppm");

        PPM_exportSoA(output_name.c_str(), outputImage);

        image_delete(outputImage);
        image_delete(inputImage);
    }

    free(kernel);
    return 0;
}