#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <string>
#include <cassert>

#include "image/PpmParser.h"
#include "kernel/Kernel.h"
#include "convolution/Convolution.cuh"

#define IMPORT_PATH "../resources/source/"
#define EXPORT_PATH "../resources/results/"
#define IMAGE "lake"

#define BLOCK_WIDTH_NAIVE 32

#define BLOCK_WIDTH (TILE_WIDTH)
static_assert(BLOCK_WIDTH * BLOCK_WIDTH <= 1024, "max number of threads per block exceeded");

#define ITER 1

#define SOA true
#define NO_PADDING true

#define NAIVE false
#define CONSTANT_MEMORY false
#define TILING true

__constant__ float MASK[MASK_WIDTH * MASK_WIDTH];


static void CheckCudaErrorAux(const char *, unsigned, const char *,
                              hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
                              const char *statement, hipError_t err) {
    if (err == hipSuccess)
        return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "("
              << err << ") at " << file << ":" << line << std::endl;
    exit(1);
}

int main() {
    assert(MASK_WIDTH == 3);

    std::string filename;
    std::string output_name;

    std::string log;
    log.append("Cuda version ");

    filename.append(IMPORT_PATH).append(IMAGE).append(".ppm");
    output_name.append(EXPORT_PATH).append(IMAGE).append("Cuda");

    float *kernel = createKernel(kernelsType::outline);

    float time = 0;

    std::chrono::high_resolution_clock::time_point startTime;
    std::chrono::high_resolution_clock::time_point endTime;

    if (!SOA) {
        output_name.append("AoS");
        log.append("AoS ");

        Image_t* inputImage = PPM_import(filename.c_str());

        int imageWidth = image_getWidth(inputImage);
        int imageHeight = image_getHeight(inputImage);
        int imageChannels = image_getChannels(inputImage);

        Image_t* outputImage;

        float *imageData;
        float *outputData;

        if (!NO_PADDING) {
            if (TILING) {
                log.append("with tiling ");
                output_name.append("Tiling");

                float *device_imageData;
                float *device_outputData;

                for (int i = 0; i < ITER; i++) {
                    if (i != 0) image_delete(outputImage);

                    outputImage = new_image(imageWidth, imageHeight, imageChannels);

                    imageData = image_getData(inputImage);
                    outputData = image_getData(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageData, imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH),
                                 ceil((float) imageHeight / BLOCK_WIDTH));
                    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);

                    convolutionTiling<<<dimGrid, dimBlock>>>(device_imageData,
                                                             device_outputData, imageWidth, imageHeight, imageChannels);

                    CUDA_CHECK_RETURN(hipMemcpy(outputData, device_outputData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                    hipFree(device_imageData);
                    hipFree(device_outputData);
                }
            }

            if (CONSTANT_MEMORY) {
                log.append("with constant memory ");
                output_name.append("ConstantMemory");

                float *device_imageData;
                float *device_outputData;

                for (int i = 0; i < ITER; i++) {

                    if (i != 0) image_delete(outputImage);

                    outputImage = new_image(imageWidth, imageHeight, imageChannels);

                    imageData = image_getData(inputImage);
                    outputData = image_getData(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageData, imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                                 ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                    dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                    convolutionConstantMemory<<<dimGrid, dimBlock>>>(device_imageData,
                                                                     device_outputData, imageWidth, imageHeight,
                                                                     imageChannels);

                    CUDA_CHECK_RETURN(hipDeviceSynchronize());

                    CUDA_CHECK_RETURN(hipMemcpy(outputData, device_outputData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                    hipFree(device_imageData);
                    hipFree(device_outputData);
                }
            }

            if (NAIVE) {
                log.append("naive ");
                output_name.append("Naive");

                float *device_imageData;
                float *device_outputData;
                float *device_maskData;

                for (int i = 0; i < ITER; i++) {
                    if (i != 0) image_delete(outputImage);

                    outputImage = new_image(imageWidth, imageHeight, imageChannels);

                    imageData = image_getData(inputImage);
                    outputData = image_getData(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_maskData,
                                                 MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMemcpy(device_imageData, imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_maskData, kernel,
                                                 MASK_WIDTH * MASK_WIDTH * sizeof(float),
                                                 hipMemcpyHostToDevice));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                                 ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                    dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                    convolutionNaive<<<dimGrid, dimBlock>>>(device_imageData, device_maskData,
                                                            device_outputData, imageWidth, imageHeight, imageChannels);

                    CUDA_CHECK_RETURN(hipDeviceSynchronize());

                    CUDA_CHECK_RETURN(hipMemcpy(outputData, device_outputData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                    hipFree(device_imageData);
                    hipFree(device_outputData);
                    hipFree(device_maskData);
                }
            }
        }

        if (NO_PADDING) {
            log.append("no padding ");
            output_name.append("NoPadding");

            int outputWidth = imageWidth - 2;
            int outputHeight = imageHeight - 2;

            if (TILING) {
                log.append("with tiling ");
                output_name.append("Tiling");

                float *device_imageData;
                float *device_outputData;

                for (int i = 0; i < ITER; i++) {
                    if (i != 0) image_delete(outputImage);

                    outputImage = new_image(outputWidth, outputHeight, imageChannels);

                    imageData = image_getData(inputImage);
                    outputData = image_getData(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputData,
                                                 outputWidth * outputHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageData, imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH),
                                 ceil((float) imageHeight / BLOCK_WIDTH));
                    dim3 dimBlock(BLOCK_WIDTH, BLOCK_WIDTH);

                    convolutionTilingNoPadding<<<dimGrid, dimBlock>>>(device_imageData,
                                                                      device_outputData, imageWidth, imageHeight,
                                                                      imageChannels);

                    CUDA_CHECK_RETURN(hipMemcpy(outputData, device_outputData,
                                                 outputWidth * outputHeight * imageChannels * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                    hipFree(device_imageData);
                    hipFree(device_outputData);
                }
            }

            if (CONSTANT_MEMORY) {
                log.append("naive with constant memory ");
                output_name.append("ConstantMemory");

                float *device_imageData;
                float *device_outputData;

                for (int i = 0; i < ITER; i++) {

                    if (i != 0) image_delete(outputImage);

                    outputImage = new_image(outputWidth, outputHeight, imageChannels);

                    imageData = image_getData(inputImage);
                    outputData = image_getData(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputData,
                                                 outputWidth * outputHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageData, imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                                 ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                    dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                    convolutionConstantMemoryNoPadding<<<dimGrid, dimBlock>>>(device_imageData,
                                                                              device_outputData, imageWidth,
                                                                              imageHeight, imageChannels);

                    CUDA_CHECK_RETURN(hipDeviceSynchronize());

                    CUDA_CHECK_RETURN(hipMemcpy(outputData, device_outputData,
                                                 outputWidth * outputHeight * imageChannels * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                    hipFree(device_imageData);
                    hipFree(device_outputData);
                }
            }

            if (NAIVE) {
                log.append("naive ");

                float *device_imageData;
                float *device_outputData;
                float *device_maskData;

                for (int i = 0; i < ITER; i++) {
                    if (i != 0) image_delete(outputImage);

                    outputImage = new_image(outputWidth, outputHeight, imageChannels);

                    imageData = image_getData(inputImage);
                    outputData = image_getData(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputData,
                                                 outputWidth * outputHeight * imageChannels * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_maskData,
                                                 MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMemcpy(device_imageData, imageData,
                                                 imageWidth * imageHeight * imageChannels * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_maskData, kernel,
                                                 MASK_WIDTH * MASK_WIDTH * sizeof(float),
                                                 hipMemcpyHostToDevice));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                                 ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                    dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                    convolutionNaiveNoPadding<<<dimGrid, dimBlock>>>(device_imageData, device_maskData,
                                                                     device_outputData, imageWidth, imageHeight,
                                                                     imageChannels);

                    CUDA_CHECK_RETURN(hipDeviceSynchronize());

                    CUDA_CHECK_RETURN(hipMemcpy(outputData, device_outputData,
                                                 outputWidth * outputHeight * imageChannels * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                    hipFree(device_imageData);
                    hipFree(device_outputData);
                    hipFree(device_maskData);
                }
            }
        }
        log.append("took ").append(std::to_string(time/ITER)).append(" seconds");
        printf("%s\n", log.c_str());

        output_name.append(".ppm");

        PPM_export(output_name.c_str(), outputImage);

        image_delete(outputImage);
        image_delete(inputImage);
    }

    if(SOA) {
        output_name.append("SoA");
        log.append("SoA ");

        ImageSoA_t* inputImage = PPM_importSoA(filename.c_str());

        int imageWidth = image_getWidth(inputImage);
        int imageHeight = image_getHeight(inputImage);
        int imageChannels = image_getChannels(inputImage);

        assert(imageChannels == 3);

        float *imageDataR;
        float *imageDataG;
        float *imageDataB;

        ImageSoA_t* outputImage;

        float *outputDataR;
        float *outputDataG;
        float *outputDataB;

        float *device_imageDataR;
        float *device_imageDataG;
        float *device_imageDataB;

        float *device_outputDataR;
        float *device_outputDataG;
        float *device_outputDataB;

        if (!NO_PADDING) {
            if (NAIVE) {
                log.append("naive ");
                output_name.append("Naive");

                float *device_maskData;

                for (int i = 0; i < ITER; i++) {
                    if (i != 0) image_delete(outputImage);

                    outputImage = new_imageSoA(imageWidth, imageHeight, imageChannels);

                    imageDataR = image_getR(inputImage);
                    imageDataG = image_getG(inputImage);
                    imageDataB = image_getB(inputImage);

                    outputDataR = image_getR(outputImage);
                    outputDataG = image_getG(outputImage);
                    outputDataB = image_getB(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_maskData,
                                                 MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataR, imageDataR,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataG, imageDataG,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataB, imageDataB,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));

                    CUDA_CHECK_RETURN(hipMemcpy(device_maskData, kernel,
                                                 MASK_WIDTH * MASK_WIDTH * sizeof(float),
                                                 hipMemcpyHostToDevice));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                                 ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                    dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                    convolutionNaiveSoA<<<dimGrid, dimBlock>>>(device_imageDataR, device_imageDataG, device_imageDataB,
                                                               device_maskData, device_outputDataR, device_outputDataG,
                                                               device_outputDataB,
                                                               imageWidth, imageHeight);

                    CUDA_CHECK_RETURN(hipDeviceSynchronize());

                    CUDA_CHECK_RETURN(hipMemcpy(outputDataR, device_outputDataR,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataG, device_outputDataG,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataB, device_outputDataB,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                    hipFree(device_maskData);
                }
            }

            if (CONSTANT_MEMORY) {
                log.append("constant memory ");
                output_name.append("ConstantMemory");

                for (int i = 0; i < ITER; i++) {
                    if (i != 0) image_delete(outputImage);

                    outputImage = new_imageSoA(imageWidth, imageHeight, imageChannels);

                    imageDataR = image_getR(inputImage);
                    imageDataG = image_getG(inputImage);
                    imageDataB = image_getB(inputImage);

                    outputDataR = image_getR(outputImage);
                    outputDataG = image_getG(outputImage);
                    outputDataB = image_getB(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataR, imageDataR,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataG, imageDataG,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataB, imageDataB,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));

                    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                                 ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                    dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                    convolutionConstantMemorySoA<<<dimGrid, dimBlock>>>(device_imageDataR, device_imageDataG,
                                                                        device_imageDataB,
                                                                        device_outputDataR, device_outputDataG,
                                                                        device_outputDataB,
                                                                        imageWidth, imageHeight);

                    CUDA_CHECK_RETURN(hipDeviceSynchronize());

                    CUDA_CHECK_RETURN(hipMemcpy(outputDataR, device_outputDataR,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataG, device_outputDataG,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataB, device_outputDataB,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();
                }
            }

            if (TILING) {
                log.append("tiling ");
                output_name.append("Tiling");

                for (int i = 0; i < ITER; i++) {
                    if (i != 0) image_delete(outputImage);

                    outputImage = new_imageSoA(imageWidth, imageHeight, imageChannels);

                    imageDataR = image_getR(inputImage);
                    imageDataG = image_getG(inputImage);
                    imageDataB = image_getB(inputImage);

                    outputDataR = image_getR(outputImage);
                    outputDataG = image_getG(outputImage);
                    outputDataB = image_getB(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataR, imageDataR,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataG, imageDataG,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataB, imageDataB,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));

                    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                                 ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                    dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                    convolutionTilingSoA<<<dimGrid, dimBlock>>>(device_imageDataR, device_imageDataG, device_imageDataB,
                                                                device_outputDataR, device_outputDataG,
                                                                device_outputDataB,
                                                                imageWidth, imageHeight);

                    CUDA_CHECK_RETURN(hipMemcpy(outputDataR, device_outputDataR,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataG, device_outputDataG,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataB, device_outputDataB,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();
                }
            }
        }

        if (NO_PADDING) {
            log.append("no padding ");
            output_name.append("NoPadding");

            int outputWidth = imageWidth - 2;
            int outputHeight = imageHeight - 2;

            if (NAIVE) {
                log.append("naive ");
                output_name.append("Naive");

                float *device_maskData;

                for (int i = 0; i < ITER; i++) {
                    if (i != 0) image_delete(outputImage);

                    outputImage = new_imageSoA(outputWidth, outputHeight, imageChannels);

                    imageDataR = image_getR(inputImage);
                    imageDataG = image_getG(inputImage);
                    imageDataB = image_getB(inputImage);

                    outputDataR = image_getR(outputImage);
                    outputDataG = image_getG(outputImage);
                    outputDataB = image_getB(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataR,
                                                 outputWidth * outputHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataG,
                                                 outputWidth * outputHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataB,
                                                 outputWidth * outputHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_maskData,
                                                 MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataR, imageDataR,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataG, imageDataG,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataB, imageDataB,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));

                    CUDA_CHECK_RETURN(hipMemcpy(device_maskData, kernel,
                                                 MASK_WIDTH * MASK_WIDTH * sizeof(float),
                                                 hipMemcpyHostToDevice));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                                 ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                    dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                    convolutionNaiveNoPaddingSoA<<<dimGrid, dimBlock>>>(device_imageDataR, device_imageDataG, device_imageDataB,
                                                               device_maskData, device_outputDataR, device_outputDataG,
                                                               device_outputDataB,
                                                               imageWidth, imageHeight);

                    CUDA_CHECK_RETURN(hipDeviceSynchronize());

                    CUDA_CHECK_RETURN(hipMemcpy(outputDataR, device_outputDataR,
                                                 outputWidth * outputHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataG, device_outputDataG,
                                                 outputWidth * outputHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataB, device_outputDataB,
                                                 outputWidth * outputHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();

                    hipFree(device_maskData);
                }
            }

            if (CONSTANT_MEMORY) {
                log.append("constant memory ");
                output_name.append("ConstantMemory");

                for (int i = 0; i < ITER; i++) {
                    if (i != 0) image_delete(outputImage);

                    outputImage = new_imageSoA(outputWidth, outputHeight, imageChannels);

                    imageDataR = image_getR(inputImage);
                    imageDataG = image_getG(inputImage);
                    imageDataB = image_getB(inputImage);

                    outputDataR = image_getR(outputImage);
                    outputDataG = image_getG(outputImage);
                    outputDataB = image_getB(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataR,
                                                 outputWidth * outputHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataG,
                                                 outputWidth * outputHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataB,
                                                 outputWidth * outputHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataR, imageDataR,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataG, imageDataG,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataB, imageDataB,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));

                    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                                 ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                    dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                    convolutionConstantMemoryNoPaddingSoA<<<dimGrid, dimBlock>>>(device_imageDataR, device_imageDataG,
                                                                        device_imageDataB,
                                                                        device_outputDataR, device_outputDataG,
                                                                        device_outputDataB,
                                                                        imageWidth, imageHeight);

                    CUDA_CHECK_RETURN(hipDeviceSynchronize());

                    CUDA_CHECK_RETURN(hipMemcpy(outputDataR, device_outputDataR,
                                                 outputWidth * outputHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataG, device_outputDataG,
                                                 outputWidth * outputHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataB, device_outputDataB,
                                                 outputWidth * outputHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();
                }
            }

            if (TILING) {
                log.append("tiling ");
                output_name.append("Tiling");

                for (int i = 0; i < ITER; i++) {
                    if (i != 0) image_delete(outputImage);

                    outputImage = new_imageSoA(outputWidth, outputHeight, imageChannels);

                    imageDataR = image_getR(inputImage);
                    imageDataG = image_getG(inputImage);
                    imageDataB = image_getB(inputImage);

                    outputDataR = image_getR(outputImage);
                    outputDataG = image_getG(outputImage);
                    outputDataB = image_getB(outputImage);

                    startTime = std::chrono::high_resolution_clock::now();

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataR,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataG,
                                                 imageWidth * imageHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageDataB,
                                                 imageWidth * imageHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataR,
                                                 outputWidth * outputHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataG,
                                                 outputWidth * outputHeight * sizeof(float)));
                    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputDataB,
                                                 outputWidth * outputHeight * sizeof(float)));

                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataR, imageDataR,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataG, imageDataG,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));
                    CUDA_CHECK_RETURN(hipMemcpy(device_imageDataB, imageDataB,
                                                 imageWidth * imageHeight * sizeof(float),
                                                 hipMemcpyHostToDevice));

                    CUDA_CHECK_RETURN(hipMemcpyToSymbol(HIP_SYMBOL(MASK), kernel, MASK_WIDTH * MASK_WIDTH * sizeof(float)));

                    dim3 dimGrid(ceil((float) imageWidth / BLOCK_WIDTH_NAIVE),
                                 ceil((float) imageHeight / BLOCK_WIDTH_NAIVE));
                    dim3 dimBlock(BLOCK_WIDTH_NAIVE, BLOCK_WIDTH_NAIVE);

                    convolutionTilingNoPaddingSoA<<<dimGrid, dimBlock>>>(device_imageDataR, device_imageDataG, device_imageDataB,
                                                                device_outputDataR, device_outputDataG,
                                                                device_outputDataB,
                                                                imageWidth, imageHeight);

                    CUDA_CHECK_RETURN(hipMemcpy(outputDataR, device_outputDataR,
                                                 outputWidth * outputHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataG, device_outputDataG,
                                                 outputWidth * outputHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));
                    CUDA_CHECK_RETURN(hipMemcpy(outputDataB, device_outputDataB,
                                                 outputWidth * outputHeight * sizeof(float),
                                                 hipMemcpyDeviceToHost));

                    endTime = std::chrono::high_resolution_clock::now();
                    time += std::chrono::duration_cast<std::chrono::duration<float>>(endTime - startTime).count();
                }
            }
        }

        log.append("took ").append(std::to_string(time/ITER)).append(" seconds");
        printf("%s\n", log.c_str());

        output_name.append(".ppm");

        PPM_exportSoA(output_name.c_str(), outputImage);

        hipFree(device_imageDataR);
        hipFree(device_imageDataG);
        hipFree(device_imageDataB);
        hipFree(device_outputDataR);
        hipFree(device_outputDataG);
        hipFree(device_outputDataB);

        image_delete(outputImage);
        image_delete(inputImage);
    }

    free(kernel);
    return 0;
}