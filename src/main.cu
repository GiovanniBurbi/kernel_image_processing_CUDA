#include "hip/hip_runtime.h"
#include <iostream>
#include "image/PpmParser.h"
#include "kernel/Kernel.h"
#include "convolution/Convolution.cuh"

#define IMPORT_PATH "../resources/source/"
#define EXPORT_PATH "../resources/results/"
#define IMAGE "lake"

static void CheckCudaErrorAux(const char *, unsigned, const char *,
                              hipError_t);
#define CUDA_CHECK_RETURN(value) CheckCudaErrorAux(__FILE__,__LINE__, #value, value)

/**
 * Check the return value of the CUDA runtime API call and exit
 * the application if the call has failed.
 */
static void CheckCudaErrorAux(const char *file, unsigned line,
                              const char *statement, hipError_t err) {
    if (err == hipSuccess)
        return;
    std::cerr << statement << " returned " << hipGetErrorString(err) << "("
              << err << ") at " << file << ":" << line << std::endl;
    exit(1);
}


int main() {
    std::string filename;
    std::string output_name;

    filename.append(IMPORT_PATH).append(IMAGE).append(".ppm");
    output_name.append(EXPORT_PATH).append(IMAGE);

    float* kernel = createKernel(kernelsType::outline);

    Image_t* image = PPM_import(filename.c_str());

    int width = image_getWidth(image);
    int height = image_getHeight(image);
    int channels = image_getChannels(image);

    int outputWidth = width - MASK_RADIUS * 2;
    int outputHeight = height - MASK_RADIUS * 2;

    Image_t* output = new_image(outputWidth, outputHeight, channels);

    float* host_imageData = image_getData(image);
    float * host_outputData = image_getData(output);

    float *device_imageData;
    float *device_outputData;
    float *device_maskData;

    CUDA_CHECK_RETURN(hipMalloc((void **) &device_imageData,
               width * height * channels * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &device_outputData,
               outputWidth * outputHeight * channels * sizeof(float)));
    CUDA_CHECK_RETURN(hipMalloc((void **) &device_maskData,
               MASK_WIDTH * MASK_WIDTH * sizeof(float)));

    CUDA_CHECK_RETURN(hipMemcpy(device_imageData, host_imageData,
               width * height * channels * sizeof(float),
               hipMemcpyHostToDevice));
    CUDA_CHECK_RETURN(hipMemcpy(device_maskData, kernel,
               MASK_WIDTH * MASK_WIDTH * sizeof(float),
               hipMemcpyHostToDevice));

    dim3 dimBlock(32, 32);
    dim3 dimGrid(ceil((float)outputWidth / dimBlock.x), ceil((float)outputHeight / dimBlock.y));

    convolutionNaive<<<dimGrid, dimBlock>>>(device_imageData, device_maskData,
                                            device_outputData, width, height, channels);

    CUDA_CHECK_RETURN(hipDeviceSynchronize());

    CUDA_CHECK_RETURN(hipMemcpy(host_outputData, device_outputData,
               outputWidth * outputHeight * channels * sizeof(float),
               hipMemcpyDeviceToHost));

    output_name.append(".ppm");

    PPM_export(output_name.c_str(), output);

    hipFree(device_imageData);
    hipFree(device_outputData);
    hipFree(device_maskData);

    image_delete(image);
    image_delete(output);
    free(kernel);

    return 0;
}
